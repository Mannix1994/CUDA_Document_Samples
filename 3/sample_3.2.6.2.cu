#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#ifndef NDEBUG
#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))
#else
#define CHECK_STATUS(status) status
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void MyKernel(float* data){

}

int main(int argc, char **argv) {

    size_t size = 1024 * sizeof(float);
    CHECK_STATUS(hipSetDevice(0));     // 选择设备0
    float* p0;
    CHECK_STATUS(hipMalloc(&p0, size));// 在设备0上分配设备内存
    MyKernel<<<1000, 128>>>(p0);        // 在设备0上运行MyKernel
    CHECK_STATUS(hipGetLastError());

    CHECK_STATUS(hipSetDevice(1));     // 选择设备1
    float* p1;
    CHECK_STATUS(hipMalloc(&p1, size));// 在设备1上分配设备内存
    MyKernel<<<1000, 128>>>(p1);        // 在设备1上运行MyKernel
    CHECK_STATUS(hipGetLastError());

    CHECK_STATUS(hipFree(p0));
    CHECK_STATUS(hipFree(p1));
    return 0;
}
