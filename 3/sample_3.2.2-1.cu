#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


static const int N = 512;

#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))

//
__global__ void VecAdd(float *A, float *B, float *C,int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
    CHECK_STATUS(hipSetDevice(0));
    float a[N];
    float b[N];
    for(int i = 0;i<N;i++){
        a[i] = i;
        b[i] = i;
    }
    float c[N];

    float *d_a,*d_b,*d_c;

    //分配显存
    CHECK_STATUS(hipMalloc(&d_a, N*sizeof(float)));
    CHECK_STATUS(hipMalloc(&d_b, N*sizeof(float)));
    CHECK_STATUS(hipMalloc(&d_c, N*sizeof(float)));

    // 把数据从内存复制到显存
    CHECK_STATUS(hipMemcpy(d_a,a,N* sizeof(float),hipMemcpyHostToDevice));
    CHECK_STATUS(hipMemcpy(d_b,b,N* sizeof(float),hipMemcpyHostToDevice));

    // 调用kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

    // 检查错误
    CHECK_STATUS(hipGetLastError());

    // 从显存把数据复制到内存
    CHECK_STATUS(hipMemcpy(c,d_c,N* sizeof(float),hipMemcpyDeviceToHost));

    // 打印
    for(int i=0;i<N;i++){
        printf("%f\t",c[i]);
        if(i%50==0 && i>0)
            printf("\n");
    }
    printf("\n");
    
    //释放显存
    CHECK_STATUS(hipFree(d_a));
    CHECK_STATUS(hipFree(d_b));
    CHECK_STATUS(hipFree(d_c));
    return 0;
}
