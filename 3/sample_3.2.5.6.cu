#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#ifndef NDEBUG
#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))
#else
#define CHECK_STATUS(status) status
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void MyKernel(float* output, float* input, size_t size){
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i>size/ sizeof(float))
        return;
    output[i] = input[i] - 5;
}

int main(int argc, char **argv) {
    CHECK_STATUS(hipSetDevice(0));

    // 创建两个流
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        CHECK_STATUS(hipStreamCreate(&stream[i]));

    // 在主机内存上分配页锁存(page-locked memory)
    const int N = 512;
    size_t size = N * sizeof(float);
    float *hostPtr;
    CHECK_STATUS(hipHostMalloc(&hostPtr, 2 * size, hipHostMallocDefault, hipHostMallocDefault));
    // 初始化
    for(size_t i=0;i<N*2;i++)
        hostPtr[i] = i;

    // 分配设备内存
    float *inputDevPtr,*outputDevPtr;
    CHECK_STATUS(hipMalloc(&inputDevPtr,2*size));
    CHECK_STATUS(hipMalloc(&outputDevPtr,2*size));

    // 创建事件
    hipEvent_t start, stop;
    CHECK_STATUS(hipEventCreate(&start));
    CHECK_STATUS(hipEventCreate(&stop));

    // 录制事件
    CHECK_STATUS(hipEventRecord(start, 0));
    for (int i = 0; i < 2; ++i) {
        // 把数据从页锁存复制到设备内存
        CHECK_STATUS(hipMemcpyAsync(inputDevPtr + i * N, hostPtr + i * N,
                                     size, hipMemcpyHostToDevice, stream[i]));
        // 调用kernel
        MyKernel<<<100, 512, 0, stream[i]>>>(outputDevPtr + i * N, inputDevPtr + i * N, size);
        // 检查错误
        CHECK_STATUS(hipGetLastError());
        // 把数据从设备内存拷贝会主机内存
        CHECK_STATUS(hipMemcpyAsync(hostPtr + i * N, outputDevPtr + i * N,
                                     size, hipMemcpyDeviceToHost, stream[i]));
    }
    // 录制事件
    CHECK_STATUS(hipEventRecord(stop, 0));

    // 调用这个函数之后，stop之前所有的cuda调用完成之后才会返回
    CHECK_STATUS(hipEventSynchronize(stop));

    // 计算运行CUDA的时间
    float elapsedTime;
    CHECK_STATUS(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("elapsedTime: %fms\n",elapsedTime);

    // 打印数据
    for(size_t i=0;i<10;i++)
    {
        printf("%.2f\t",hostPtr[i]);
    }

    // 销毁流
    for (int i = 0; i < 2; ++i)
        CHECK_STATUS(hipStreamDestroy(stream[i]));

    // 释放设备内存
    CHECK_STATUS(hipHostFree(hostPtr));
    CHECK_STATUS(hipFree(inputDevPtr));
    CHECK_STATUS(hipFree(outputDevPtr));

    // 销毁事件
    CHECK_STATUS(hipEventDestroy(start));
    CHECK_STATUS(hipEventDestroy(stop));
    return 0;
}
