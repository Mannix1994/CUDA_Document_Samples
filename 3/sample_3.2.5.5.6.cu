#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#ifndef NDEBUG
#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))
#else
#define CHECK_STATUS(status) status
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void MyKernel(float* output, float* input, size_t size){
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i>size/ sizeof(float))
        return;
    output[i] = input[i] - 5;
}

// stream回调
void CUDART_CB MyCallback(hipStream_t stream, hipError_t status, void *data){
    printf("Inside MyCallback %lu\n", (size_t)data);
}

int main(int argc, char **argv) {
    CHECK_STATUS(hipSetDevice(0));
    hipDeviceProp_t prop;
    CHECK_STATUS(hipGetDeviceProperties(&prop,0));
    printf("Kernel并发执行:%d, 异步引擎数量:%d\n",prop.concurrentKernels,prop.asyncEngineCount);

    // 创建两个流
    hipStream_t stream[2];
    for (int i = 0; i < 2; ++i)
        CHECK_STATUS(hipStreamCreate(&stream[i]));

    // 在主机内存上分配页锁存(page-locked memory)
    const int N = 512;
    size_t size = N * sizeof(float);
    float *hostPtr[2];
    for (int i = 0; i < 2; ++i)
        CHECK_STATUS(hipHostMalloc(&(hostPtr[i]), size, hipHostMallocDefault, hipHostMallocDefault));
    // 初始化
    for (int i = 0; i < 2; ++i)
        for(int j=0;j<N;j++)
            hostPtr[i][j] = j;

    // 分配设备内存
    float *inputDevPtr[2],*outputDevPtr[2];
    for (int i = 0; i < 2; ++i){
        CHECK_STATUS(hipMalloc(&(inputDevPtr[i]), size));
        CHECK_STATUS(hipMalloc(&(outputDevPtr[i]), size));
    }

    for (int i = 0; i < 2; ++i) {
        // 把数据从页锁存复制到设备内存
        CHECK_STATUS(hipMemcpyAsync(inputDevPtr[i], hostPtr[i],
                                     size, hipMemcpyHostToDevice, stream[i]));
        // 调用kernel
        MyKernel<<<100, 512, 0, stream[i]>>>(outputDevPtr[i], inputDevPtr[i], size);
        // 检查错误
        CHECK_STATUS(hipGetLastError());
        // 把数据从设备内存拷贝会主机内存
        CHECK_STATUS(hipMemcpyAsync(hostPtr[i], outputDevPtr[i],
                                     size, hipMemcpyDeviceToHost, stream[i]));

        // 添加回调。MyCallback会在之前添加到流的任务完成以后被调用
        // 不能在回调里面调用CUDA API，避免造成死锁
        CHECK_STATUS(hipStreamAddCallback(stream[i], MyCallback, (void*)i, 0));
    }

    // 等待所有流中的的所有任务完成
    hipDeviceSynchronize();

    // 打印数据
    for(size_t i=0;i<10;i++)
    {
        printf("%.2f\t",hostPtr[0][i]);
    }

    // 销毁流
    for (int i = 0; i < 2; ++i)
        CHECK_STATUS(hipStreamDestroy(stream[i]));

    // 释放设备内存
    for (int i = 0; i < 2; ++i)
    {
        CHECK_STATUS(hipHostFree(hostPtr[i]));
        CHECK_STATUS(hipFree(inputDevPtr[i]));
        CHECK_STATUS(hipFree(outputDevPtr[i]));
    }
    return 0;
}
