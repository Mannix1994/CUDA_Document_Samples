#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#ifndef NDEBUG
#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))
#else
#define CHECK_STATUS(status) status
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void MyKernel(float* data){

}

int main(int argc, char **argv) {
    int can;
    CHECK_STATUS(hipDeviceCanAccessPeer(&can,0,1));
    printf("是否支持peer-to-peer access：%d",can);

    size_t size = 1024 * sizeof(float);
    CHECK_STATUS(hipSetDevice(0));     // 选择设备0
    float* p0;
    CHECK_STATUS(hipMalloc(&p0, size));// 在设备0上分配设备内存
    MyKernel<<<1000, 128>>>(p0);        // 在设备0上运行MyKernel
    CHECK_STATUS(hipGetLastError());

    CHECK_STATUS(hipSetDevice(1));     // 选择设备1
    hipDeviceEnablePeerAccess(0, 0);   // 启用peer-to-peer访问支持
    MyKernel<<<1000, 128>>>(p0);        // 在设备1上运行MyKernel，访问在设备0上的内存p0
    CHECK_STATUS(hipGetLastError());

    CHECK_STATUS(hipFree(p0));
    return 0;
}
