#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>



#ifndef NDEBUG
#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))
#else
#define CHECK_STATUS(status) status
#endif
//////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {
    int deviceCount;
    CHECK_STATUS(hipGetDeviceCount(&deviceCount));
    int device;
    for (device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        CHECK_STATUS(hipGetDeviceProperties(&deviceProp, device));
        printf("Device %d has compute capability %d.%d.\n",
               device, deviceProp.major, deviceProp.minor);
    }
    return 0;
}
