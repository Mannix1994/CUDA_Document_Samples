#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


static const int M = 5;//行
static const int N = 10;//列

#define CHECK_STATUS(status) \
    if (status != hipSuccess) \
        fprintf(stderr, "File: %s\nLine:%d Function:%s>>>%s\n", __FILE__, __LINE__, __FUNCTION__,\
        hipGetErrorString(status))

//二维数组相加
__global__ void MatAdd(float *A, float *B, float *C)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    int index = i * N + j;
    C[index] = A[index] + B[index];
}

int main(int argc, char **argv) {
    CHECK_STATUS(hipSetDevice(0));
    const int SIZE = M * N;
    float a[SIZE];
    float b[SIZE];
    for(int i = 0;i<SIZE;i++){
        a[i] = i;
        b[i] = i;
    }
    float c[SIZE];

    float *d_a,*d_b,*d_c;

    //分配显存
    CHECK_STATUS(hipMalloc(&d_a, SIZE*sizeof(float)));
    CHECK_STATUS(hipMalloc(&d_b, SIZE*sizeof(float)));
    CHECK_STATUS(hipMalloc(&d_c, SIZE*sizeof(float)));

    // 把数据从内存复制到显存
    CHECK_STATUS(hipMemcpy(d_a,a,SIZE* sizeof(float),hipMemcpyHostToDevice));
    CHECK_STATUS(hipMemcpy(d_b,b,SIZE* sizeof(float),hipMemcpyHostToDevice));

    // 调用kernel
    int numBlocks = 1;
    dim3 threadsPerBlock(M, N);
    MatAdd<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c);

    // 检查错误
    CHECK_STATUS(hipGetLastError());

    // 从显存把数据复制到内存
    CHECK_STATUS(hipMemcpy(c,d_c,SIZE* sizeof(float),hipMemcpyDeviceToHost));

    // 打印
    for(int i=0;i<M;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f\t",c[i*N + j]);
        printf("\n");
    }

    //释放显存
    CHECK_STATUS(hipFree(d_a));
    CHECK_STATUS(hipFree(d_b));
    CHECK_STATUS(hipFree(d_c));
    return 0;
}
